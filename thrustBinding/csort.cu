#include <thrust/system_error.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <hip/hip_runtime.h>

// #include <algorithm>
#include <new>
#include <iostream>
#include <cstdlib>

#ifdef DEBUG
  size_t deviceFree()
  {
    const size_t MB = 1<<20;

    size_t reserved, total;
    hipMemGetInfo( &reserved, &total );
    void* pool;
    while( hipMalloc( (void**)&pool, reserved ) != hipSuccess )
    {
        reserved -= MB;
        if( reserved < MB )
        {
            break;
        }
    }
    hipFree(pool);
    return reserved;
  }

  size_t deviceFreeThrust()
  {
    const size_t MB = 1<<20;

    size_t reserved, total;
    hipMemGetInfo( &reserved, &total );
    bool failed = true;
    thrust::device_vector<int> d_vec;
    while( failed )
    {
        try
        {
            d_vec.resize( reserved/sizeof(int) );
            //std::cout << reserved/sizeof(int) << std::endl;
            failed = false;
        }
        catch (...)
        {
        }
        reserved -= MB;
        if( reserved < MB )
        {
            break;
        }
    }
    d_vec.clear();
    return reserved;
  }
#endif


#ifdef  __cplusplus
extern "C" {
#endif

  //Sort for integer value, double key arrays
  void sort_int_by_key_wrapper( int& N, double* keys, int* values )
  {
#ifdef DEBUG
    size_t free, total;
    hipMemGetInfo( &free , &total );
    free = deviceFreeThrust();

    // allocate and sort need two times the data memory
    const size_t memReq = N * sizeof( int ) * 2 +
                          N * sizeof( double ) * 2;
    if( free <= memReq )
    {
      std::cerr << "Not enough memory on your GPU!\n"
                << "You need at least      : " << memReq /1024/1024
                << "MiB\nof free global memory to sort this dataset.\n"
                << "Note: If you can, close your XSession to\n"
                << "      free some GPU memory.\n"
                << "      Run 'nvidia-smi' for further information."
                << std::endl;
      std::cerr << "Free Memory            : " << free/1024/1024  << "MiB" << std::endl; 
      std::cerr << "Total amount of memory : " << total/1024/1024 << "MiB" << std::endl;
      exit(-1);
    }
#endif

    thrust::device_vector<int> d_values;
    thrust::device_vector<double> d_keys;
    // transfer data to the device
    try
    {
      d_values.resize( N );
      d_keys.resize(   N );
      thrust::copy( values, values + N, d_values.begin() );
      thrust::copy( keys,   keys + N,   d_keys.begin()   );
    }
    catch(std::bad_alloc &e)
    {
      std::cerr << "Couldn't allocate device vectors" << std::endl;
      size_t free, total;
      hipMemGetInfo (&free , &total);
      std::cerr << "Tried to allocate      : " << ( sizeof( int ) * N + sizeof( double ) * N )
                                                  /1024/1024 << "MiB" << std::endl;
      std::cerr << "Free Memory            : " << free/1024/1024  << "MiB" << std::endl;
      std::cerr << "Total amount of memory : " << total/1024/1024 << "MiB" << std::endl;
      exit(-1);
    }


    // sort data on the device
    //   note: the sort function we allocate at least
    //         the same amount of temporary data again
    //         than the device vector's size in memory
    try
    {
      thrust::sort_by_key( d_keys.begin(), d_keys.end(), d_values.begin() );
                   //, thrust::less<int>());
    }
    catch(std::bad_alloc &e)
    {
      std::cerr << "Ran out of memory while sorting" << std::endl;
      exit(-1);
    }
    catch(thrust::system_error &e)
    {
      std::cerr << "Some other error happened during sort: " << e.what() << std::endl;
      exit(-1);
    }

    // copy data back to host array
    thrust::copy(d_values.begin(), d_values.end(), values );
    thrust::copy(d_keys.begin(),   d_keys.end(),   keys   );

  }

  //Sort for integer arrays
  void sort_int_wrapper( int& N, int *data )
  {
#ifdef DEBUG
    size_t free, total;
    hipMemGetInfo( &free , &total );
    free = deviceFreeThrust();

    // allocate and sort need two times the data memory
    if( free / sizeof( int ) / 2 <= N )
    {
      std::cerr << "Not enough memory on your GPU!\n"
                << "You need at least      : " << 2 * sizeof( int ) * N /1024/1024
                << "MiB\nof free global memory to sort this dataset.\n"
                << "Note: If you can, close your XSession to\n"
                << "      free some GPU memory.\n"
                << "      Run 'nvidia-smi' for further information."
                << std::endl;
      std::cerr << "Free Memory            : " << free/1024/1024  << "MiB" << std::endl; 
      std::cerr << "Total amount of memory : " << total/1024/1024 << "MiB" << std::endl;
      exit(-1);
    }
#endif

    thrust::device_vector<int> d_vec;
    // transfer data to the device
    try
    {
      d_vec.resize( N );
      thrust::copy( data, data + N, d_vec.begin());
    }
    catch(std::bad_alloc &e)
    {
      std::cerr << "Couldn't allocate device vector" << std::endl;
      size_t free, total;
      hipMemGetInfo (&free , &total);
      std::cerr << "Tried to allocate      : " << sizeof( int ) * N /1024/1024 << "MiB" << std::endl;
      std::cerr << "Free Memory            : " << free/1024/1024  << "MiB" << std::endl;
      std::cerr << "Total amount of memory : " << total/1024/1024 << "MiB" << std::endl;
      exit(-1);
    }


    // sort data on the device
    //   note: the sort function we allocate at least
    //         the same amount of temporary data again
    //         than the device vector's size in memory
    try
    {
      thrust::sort(d_vec.begin(), d_vec.end() );
                   //, thrust::less<int>());
    }
    catch(std::bad_alloc &e)
    {
      std::cerr << "Ran out of memory while sorting" << std::endl;
      exit(-1);
    }
    catch(thrust::system_error &e)
    {
      std::cerr << "Some other error happened during sort: " << e.what() << std::endl;
      exit(-1);
    }

    // copy data back to host array
    thrust::copy(d_vec.begin(), d_vec.end(), data);

  }

#ifdef  __cplusplus
}
#endif

